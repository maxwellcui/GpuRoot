#include "hip/hip_runtime.h"
// This is the Kernal wrapper

#include "cuRoot.hh"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void printFromGPU(int N)
{
  int tid=threadIdx.x+blockDim.x*blockIdx.x;
  if(tid<N)
  {
    printf("\t\tThis is from Device block %d, thread %d, from cuRoot.\n",blockIdx.x,threadIdx.x);
  }
}

GpuInterface::GpuInterface()
:N(4)
{
  std::cout<<"\tGPU Interface constructed.\n";
}

GpuInterface::~GpuInterface()
{
  std::cout<<"\tGPU Interface deconstructed.\n";
}

void GpuInterface::printInfo()
{
  printFromGPU<<<2,2>>>(N);
  hipDeviceReset();
}
void GpuInterface::calculation()
{

}
